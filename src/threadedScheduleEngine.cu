#include "hip/hip_runtime.h"
#include<assert.h> include <cstdlib> include <cmath> include "ScheduleEngine.h" include "image.h" include 
#"coarsened_forward_convolution.h" define BLOCK1 512
using namespace std;

__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .1f*x;}

__global__ void activate_array_kernel(float *x, int n) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) x[i] = leaky_activate_kernel(x[i]); //	if (i<n){ if (x>0) x[i]= x[i] ;
 // else x[i]=.1f*x[i]; //}
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial) {
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    x[index] = (x[index] - mean[f])/(sqrtf(variance[f] + .00001f));
}



__global__ void scale_bias_kernel(float *output, float *biases, int n, int size) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}


__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size) {
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

void error(const char *s) {
	perror(s);
	assert(0);
	exit(-1);
}

dim3 cuda_gridsize(int n){
    int k = (n-1) / BLOCK1 + 1;
    int x = k;
    int y = 1;
    if(x > 65535){
        x = ceil(sqrt(k));
        y = (n-1)/(x*BLOCK1) + 1;
    }
    dim3 d = {x, y, 1};
   // printf("n=%d x=%d y=%d x*y*BLOCK1=%d\n", n, x, y, x*y*BLOCK1);
    return d;
}

void check_error(hipError_t status) {
    //hipDeviceSynchronize();
    hipError_t status2 = hipGetLastError();
    if (status != hipSuccess)
    {
        const char *s = hipGetErrorString(status);
        char buffer[256];
        printf("CUDA Error: %s\n", s);
        assert(0);
        snprintf(buffer, 256, "CUDA Error: %s", s);
        error(buffer);
    } 
    if (status2 != hipSuccess)
    {
        const char *s = hipGetErrorString(status);
        char buffer[256];
        printf("CUDA Error Prev: %s\n", s);
        assert(0);
        snprintf(buffer, 256, "CUDA Error Prev: %s", s);
        error(buffer);
    } 
}


void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial, hipStream_t stream) {
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK1, 0, stream>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}



void add_bias_gpu(float *output, float *biases, int batch, int n, int size, hipStream_t stream) {
    int num = n*size*batch;

    add_bias_kernel<<<cuda_gridsize(num), BLOCK1, 0, stream>>>(output, biases, batch, n, size);
    check_error(hipPeekAtLastError());
}


void scale_bias_gpu(float *output, float *biases, int batch, int n, int size, hipStream_t stream) {
    dim3 dimGrid((size-1)/BLOCK1 + 1, n, batch);
    dim3 dimBlock(BLOCK1, 1, 1);

    scale_bias_kernel<<<dimGrid, dimBlock, 0, stream>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

void activate_array_gpu(float *x, int n, hipStream_t stream) {
    dim3 kk=cuda_gridsize(n);
    activate_array_kernel<<<kk, BLOCK1, 0, stream>>>(x, n);
    check_error(hipPeekAtLastError());
}


void customCoarsenedConvolutionForward(float* layer_input, float* layer_output,
	// int coarsening_factor, int coarsening_stride,
	hipdnnConvolutionDescriptor_t conv_desc,
	hipdnnFilterDescriptor_t filt_desc,
	hipdnnTensorDescriptor_t input_tensor,
	float* filt){
   
	
	int pad_h, pad_w, stride_x, stride_y; // padding along h and w, vertical and horizontal stride
	int dilation_h, dilation_w; // this is 1 always for both
	hipdnnConvolutionMode_t mode;
	hipdnnDataType_t computeType;
	
	hipdnnGetConvolution2dDescriptor(
			conv_desc,
			&pad_h, &pad_w,
			&stride_x, &stride_y,
			&dilation_h, &dilation_w,
			&mode,
			&computeType);


	int k, c; // k = # of output channels, c = # of input channels
	hipdnnDataType_t datatype;
	hipdnnTensorFormat_t format;
	int kernel_h, kernel_w;
	hipdnnGetFilter4dDescriptor(
		filt_desc,
		&datatype,
		&format,
		&k, &c, &kernel_h, &kernel_w);


		hipdnnDataType_t dataType2;

	// For example, in a minibatch of RGB images, we may have
	// X[n,c,h,w], where n is the index of an image in the
	// minibatch, c is the channel (R = 0, G = 1, B = 2), and h and w
	// index a pixel (h, w) in the image (h and w are height and width)

	int batch_size, c2, input_h, input_w;
	int nStr, cStr, hStr, wStr;

	hipdnnGetTensor4dDescriptor(
		input_tensor,
		&dataType2,
		&batch_size, &c2, &input_h, &input_w,
		&nStr, &cStr, &hStr, &wStr
		);

	//cout << pad_h << endl;

	// for debugging
	int coarsening_factor = 4;
	int coarsening_stride = 32;

   if (kernel_h != kernel_w){
	   std::cout << "ERROR: Please pass a square kernel with equal height and width. Returning..." << std::endl;
	   return;
   }

   if (input_h != input_w){
		std::cout << "ERROR: Please pass a square input with equal height and width. Returning..." << std::endl;
	   return;
   }

   if (pad_h != pad_w){
	std::cout << "ERROR: Padding in both directions should be equal. Returning..." << std::endl;
	   return;
   }

   if (stride_y != stride_x || stride_y != 1){
	std::cout << "ERROR: Please ensure both stride x and stride y are equal to 1. Returning..." << std::endl;
	   return;
   }
   
   if (coarsening_stride != 32) {
	std::cout << "ERROR: Stride is not 32. This will break memory coalescing pattern. Please set stride to 32. Returning..." << 
std::endl;
	   return;
   }

   float* images = layer_input;
   float* filters = filt;
   float* output = layer_output;
   float gPadZeros = pad_h;
   int gFilterSize = kernel_h;
   int gEven = (gFilterSize % 2 == 0);
   int gInputSize = input_h;
   int gInputPlanes = c;
   
   int gNumFilters = k;

   int stride = stride_x;
   int gOutputSize = (gInputSize - gFilterSize + 2 * gPadZeros) / stride + 1;
   int gOutputSizeSquared = gOutputSize * gOutputSize;

   int gFilterSizeSquared = (gFilterSize*gFilterSize);
   int filterCubeLength = (gInputPlanes*gFilterSizeSquared);
   int gInputSizeSquared = (gInputSize * gInputSize);
	
	std::cout<<"input_h is ...."<<input_h<<std::endl;
	std::cout<<"FilterCubelength is..."<<filterCubeLength<<std::endl;
	std::cout<<"gInputSizeSquare is..."<<gInputSizeSquared<<std::endl;

   if (filterCubeLength >= 600){
	std::cout << "Allocated shared memory is not enough (filter size is too large/param::M1 in kernel generator script)." << 
std::endl;
	std::cout << "Please regenerate the kernels with large enough shared memory. Returning..." << std::endl;
	   return;
   }

   if (gInputSizeSquared >= 512){
	std::cout << "Allocated shared memory is not enough (input size is too large/param::M2 in kernel generator script)." << 
std::endl;
	std::cout << "Please regenerate the kernels with large enough shared memory. Returning..." << std::endl;
	   return;
   }

   int batchSize = 1;
   dim3 grid(batchSize * gNumFilters);
   //int nblocks = ((gOutputSizeSquared+coarsening_factor-1)/(coarsening_factor) + 31)/32 * 32;
	int nblocks = gOutputSizeSquared;
   dim3 block(nblocks);

  std::cout<<"grid"<<batchSize * gNumFilters<<endl;
  //std::cout<<"grid"<<grid;
  std::cout<<"block"<< nblocks<<endl;

   if (coarsening_factor == 1){
	   coarsened_convolution_1C32S<<<grid, block>>>(batchSize, images, filters, output, gOutputSize, gPadZeros, gEven, 
gOutputSizeSquared, gInputSize, gInputPlanes, gFilterSize, gNumFilters);
    check_error(hipPeekAtLastError());
	   return;
   }else if (coarsening_factor == 2){
	   coarsened_convolution_2C32S<<<grid, block>>>(batchSize, images, filters, output, gOutputSize, gPadZeros, gEven, 
gOutputSizeSquared, gInputSize, gInputPlanes, gFilterSize, gNumFilters);
    check_error(hipPeekAtLastError());
	   return;
   }else if (coarsening_factor == 4){
	   coarsened_convolution_4C32S<<<grid, block>>>(batchSize, images, filters, output, gOutputSize, gPadZeros, gEven, 
gOutputSizeSquared, gInputSize, gInputPlanes, gFilterSize, gNumFilters);
    check_error(hipPeekAtLastError());
	   return;
   }else if (coarsening_factor == 8){
	   coarsened_convolution_8C32S<<<grid, block>>>(batchSize, images, filters, output, gOutputSize, gPadZeros, gEven, 
gOutputSizeSquared, gInputSize, gInputPlanes, gFilterSize, gNumFilters);
    check_error(hipPeekAtLastError());
	   return;
   }else if (coarsening_factor == 16){
	   coarsened_convolution_16C32S<<<grid, block>>>(batchSize, images, filters, output, gOutputSize, gPadZeros, gEven, 
gOutputSizeSquared, gInputSize, gInputPlanes, gFilterSize, gNumFilters);
    check_error(hipPeekAtLastError());
	   return;
   }else if (coarsening_factor == 32){
	   coarsened_convolution_32C32S<<<grid, block>>>(batchSize, images, filters, output, gOutputSize, gPadZeros, gEven, 
gOutputSizeSquared, gInputSize, gInputPlanes, gFilterSize, gNumFilters);
    check_error(hipPeekAtLastError());
	   return;
   }


   std::cout << "ERROR: An invalid coarsening factor has been passed. Please ensure coarsening factor is one of 1/2/4/8/16. 
Returning..." << std::endl;
   return;
}






//functions taken from darknet frameowrk


void malloc_error() {
    fprintf(stderr, "Malloc error\n");
    exit(-1);
}

void free_node(node *n) {
	node *next;
	while(n) {
		next = n->next;
		free(n);
		n = next;
	}
}


void free_list(list *l) {
	free_node(l->front);
	free(l);
}


void list_insert(list *l, void *val) {
	node *new1 = (node *)malloc(sizeof(node));
	new1->val = val;
	new1->next = 0;

	if(!l->back){
		l->front = new1;
		new1->prev = 0;
	}else{
		l->back->next = new1;
		new1->prev = l->back;
	}
	l->back = new1;
	++l->size;
}

void file_error( const char *s) {
    fprintf(stderr, "Couldn't open file: %s\n", s);
    exit(0);
}

void strip(char *s) {
    size_t i;
    size_t len = strlen(s);
    size_t offset = 0;
    for(i = 0; i < len; ++i){
        char c = s[i];
        if(c==' '||c=='\t'||c=='\n') ++offset;
        else s[i-offset] = c;
    }
    s[len-offset] = '\0';
}

char *fgetl(FILE *fp) {
    if(feof(fp)) return 0;
    size_t size = 512;
    char *line = (char *)malloc(size*sizeof(char));
    if(!fgets(line, size, fp)){
        free(line);
        return 0;
    }

    size_t curr = strlen(line);

    while((line[curr-1] != '\n') && !feof(fp)){
        if(curr == size-1){
            size *= 2;
            line =(char *) realloc(line, size*sizeof(char));
            if(!line) {
                printf("%ld\n", size);
                malloc_error();
            }
        }
        size_t readsize = size-curr;
        if(readsize > INT_MAX) readsize = INT_MAX-1;
        fgets(&line[curr], readsize, fp);
        curr = strlen(line);
    }
    if(line[curr-1] == '\n') line[curr-1] = '\0';

    return line;
}

void **list_to_array(list *l) {
    void **a = (void **)calloc(l->size, sizeof(void*));
    int count = 0;
    node *n = l->front;
    while(n){
        a[count++] = n->val;
        n = n->next;
    }
    return a;
}

list *make_list() {
	list *l =(list *) malloc(sizeof(list));
	l->size = 0;
	l->front = 0;
	l->back = 0;
	return l;
}


list *get_paths(const char *filename) {
    char *path;
    FILE *file = fopen(filename, "r");
    if(!file) file_error(filename);
    list *lines = make_list();
    while((path=fgetl(file))){
        list_insert(lines, path);
    }
    fclose(file);
    return lines;
}

void top_k(float *a, int n, int k, int *index) {
    int i,j;
    for(j = 0; j < k; ++j) index[j] = -1;
    for(i = 0; i < n; ++i){
        int curr = i;
        for(j = 0; j < k; ++j){
            if((index[j] < 0) || a[curr] > a[index[j]]){
                int swap = curr;
                curr = index[j];
                index[j] = swap;
            }
        }
    }
}


char **get_labels(const char *filename) {
    list *plist = get_paths(filename);
    char **labels = (char **)list_to_array(plist);
    free_list(plist);
    return labels;
}




char *option_find(list *l, const char *key) {
    node *n = l->front;
    while(n){
        kvp *p = (kvp *)n->val;
        if(strcmp(p->key, key) == 0){
            p->used = 1;
            return p->val;
        }
        n = n->next;
    }
    return 0;
}
char *option_find_str(list *l, const char *key, char *def) {
    char *v = option_find(l, key);
    if(v) return v;
    if(def) fprintf(stderr, "%s: Using default '%s'\n", key, def);
    return def;
}

int option_find_int(list *l, const char *key, int def) {
    char *v = option_find(l, key);
    if(v) return atoi(v);
    fprintf(stderr, "%s: Using default '%d'\n", key, def);
    return def;
}
void option_insert(list *l, char *key, char *val) {
    kvp *p = (kvp *)malloc(sizeof(kvp));
    p->key = key;
    p->val = val;
    p->used = 0;
    list_insert(l, p);
}

int read_option(char *s, list *options) {
    size_t i;
    size_t len = strlen(s);
    char *val = 0;
    for(i = 0; i < len; ++i){
        if(s[i] == '='){
            s[i] = '\0';
            val = s+i+1;
            break;
        }
    }
    if(i == len-1) return 0;
    char *key = s;
    option_insert(options, key, val);
    return 1;
}

list *read_data_cfg(const char *filename) {
    FILE *file = fopen(filename, "r");
    if(file == 0) file_error(filename);
    char *line;
    int nu = 0;
    list *options = make_list();
    while((line=fgetl(file)) != 0){
        ++ nu;
        strip(line);
        switch(line[0]){
            case '\0':
            case '#':
            case ';':
                free(line);
                break;
            default:
                if(!read_option(line, options)){
                    fprintf(stderr, "Config file error line %d, could parse: %s\n", nu, line);
                    free(line);
                }
                break;
        }
    }
    fclose(file);
    return options;
}



void ScheduleEngine::initMutex(void){
	pthread_mutex_init(&lock, NULL);
}

void ScheduleEngine::destroyMutex(void){
	pthread_mutex_destroy(&lock);
}

void ScheduleEngine::initCond(void){
	pthread_cond_init(&cond, NULL);
}

void ScheduleEngine::destroyCond(void){
	pthread_cond_destroy(&cond);
}

ScheduleEngine::ScheduleEngine(){
	initMutex();
	initCond();
}

void ScheduleEngine:: enqueue(Operation tp){
	 pthread_mutex_lock(&lock);
	 Q.push(tp);
	 pthread_cond_signal(&cond);
	 pthread_mutex_unlock(&lock);
}

Operation ScheduleEngine::dequeue(){
	 Operation tp;
	 pthread_mutex_lock(&lock);
	 while (Q.empty()){
		printf("Wating for operations to be added in Queue\n");
		pthread_cond_wait(&cond,&lock);
	}
         tp=Q.top();
	 Q.pop();
	 pthread_mutex_unlock(&lock);
	 return (tp);
}

//Dispatch opration will execute the operation on GPU. The operation performed is based on the type of layer void 
ScheduleEngine::dispatch(Operation *tp){

	int priority=tp->priority;	// to be used later
	int i = tp->op_layer;

	NeuralNet *nm=tp->model;
	CnmemSpace space_tracker(nm->free_bytes); //need updates here //--	std::cout << "here\n"; //--	std::cout << "Free 
bytes: " << nm->free_bytes << std::endl;
	

	if(i==0){ //this is the first layer, load and resize image as per current inference pipeline
	/*	image im = load_image_color(nm->imgfname, 0, 0);
		//size? net->w in yolo
		image r = letterbox_image(im,nm->input_w, nm->input_h );
		//resize_network(net, resized.w, resized.h);
		show_image(im,"orig",5);
		show_image(r,"letterimg",5);
		//copy image data into layer_input[0]
		//memcpy(&(nm->layer_input[i]),r.data,nm->layer_input_size[i]*nm->data_type_size);
		nm->lockedcnmemMalloc(&(nm->layer_input[0]), nm->layer_input_size[0] * nm->data_type_size, NULL);*/
		space_tracker.updateSpace(CnmemSpace::SUB, nm->layer_input_size[0] * nm->data_type_size);
		//checkCudaErrors(hipMemcpy(nm->layer_input[0], r.data, nm->batch_size * nm->input_channels * nm->input_h * 
nm->input_w * nm->data_type_size, hipMemcpyHostToDevice));
	}
	float alpha = 1.0, beta = 0.0;
	float Salpha = 1.0, Sbeta = 0.0;
	double Dalpha = 1.0, Dbeta = 0.0;
	size_t cur_workspace_size;
	void *cur_workspace;

	nm->lockedcnmemMalloc(&(nm->layer_input[i + 1]), nm->layer_input_size[i+ 1] * nm->data_type_size, NULL);
	space_tracker.updateSpace(CnmemSpace::SUB, nm->layer_input_size[i + 1] * nm->data_type_size);

	if (nm->layer_type[i] == CONV) {

		// std::cout << "conv\n";
		ConvLayerParams *cur_params = (ConvLayerParams *)nm->params[i];

		cur_workspace_size = cur_params->fwd_workspace_size;
		nm->lockedcnmemMalloc(&cur_workspace, cur_workspace_size, NULL);
		// computation /* checkCUDNN(hipdnnConvolutionForward(nm->cudnn_handle, &alpha,
					cur_params->input_tensor, nm->layer_input[i],
					cur_params->filter_desc, cur_params->W,
					cur_params->conv_desc, cur_params->fwd_algo,
					cur_workspace, cur_workspace_size,
					&beta,
					cur_params->output_tensor, nm->layer_input[i + 1])); */
		//custom coarsened cuda kernel
		customCoarsenedConvolutionForward((float*)nm->layer_input[i], (float*)nm->layer_input[i+1], cur_params->conv_desc, 
cur_params->filter_desc, cur_params->input_tensor, (float*) cur_params->W);
	
	//Batch Normalization
	if(cur_params->bn==1){
		normalize_gpu((float *)nm->layer_input[i+1], (float *)cur_params->rolling_mean_gpu, (float 
*)cur_params->rolling_variance_gpu, 1 , cur_params->C_out ,cur_params->output_h*cur_params->output_w, nm->stream_compute);
		scale_bias_gpu((float *)nm->layer_input[i+1], (float *)cur_params->scales_gpu, 1, cur_params->C_out, 
cur_params->output_h*cur_params->output_w, nm->stream_compute);
		add_bias_gpu((float *)nm->layer_input[i+1], (float *)cur_params->b, 1 , cur_params->C_out, 
cur_params->output_h*cur_params->output_w, nm->stream_compute);
	}
	else{
		add_bias_gpu((float *)nm->layer_input[i+1], (float *)cur_params->b, 1 , cur_params->C_out, 
cur_params->output_h*cur_params->output_w, nm->stream_compute);
		
	}
/*-- checkCUDNN(hipdnnAddTensor(nm->cudnn_handle, &alpha,
					cur_params->bias_desc, cur_params->b,
					&alpha,
					cur_params->output_tensor, nm->layer_input[i + 1])); --*/
		// if activation required
		if (cur_params->activation_mode != ACTIVATION_NONE) {
			//Replacing cuDNN call for relu to custom leaky relu call
			float * addr= (float *)(nm->layer_input[i+1]);
			activate_array_gpu(addr, nm->layer_input_size[i+ 1],nm->stream_compute);
			
			/*checkCUDNN(hipdnnActivationForward(nm->cudnn_handle, cur_params->actv_desc,
						&alpha,
						cur_params->output_tensor, nm->layer_input[i + 1],
						&beta,
						cur_params->output_tensor, nm->layer_input[i + 1]));
			*/
		}

		space_tracker.updateSpace(CnmemSpace::SUB, cur_workspace_size);
		// std::cout << "Free bytes: " << free_bytes << std::endl;
	
	}
	else if (nm->layer_type[i] == FULLY_CONNECTED) {
		// std::cout << "FC\n";
		FCLayerParams *cur_params = (FCLayerParams *)nm->params[i];
		// std::cout << "FChere" << i << std::endl;

		if (nm->data_type == HIPDNN_DATA_FLOAT) {
			checkCUBLAS(hipblasSgemm(nm->cublas_handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,
						cur_params->C_out, nm->batch_size, cur_params->C_in,
						&Salpha,
						(float *)cur_params->W, cur_params->C_out,
						(float *)nm->layer_input[i], cur_params->C_in,
						&Sbeta,
						(float *)nm->layer_input[i + 1], cur_params->C_out));
			checkCUBLAS(hipblasSgemm(nm->cublas_handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,
						cur_params->C_out, nm->batch_size, 1,
						&Salpha,
						(float *)cur_params->b, cur_params->C_out,
						(float *)nm->one_vec, 1,
						&Salpha,
						(float *)nm->layer_input[i + 1], cur_params->C_out));
		}
		else if (nm->data_type == HIPDNN_DATA_DOUBLE) {
			checkCUBLAS(hipblasDgemm(nm->cublas_handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,
						cur_params->C_out, nm->batch_size, cur_params->C_in,
						&Dalpha,
						(double *)cur_params->W, cur_params->C_out,
						(double *)nm->layer_input[i], cur_params->C_in,
						&Dbeta,
						(double *)nm->layer_input[i + 1], cur_params->C_out));
			checkCUBLAS(hipblasDgemm(nm->cublas_handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,
						cur_params->C_out,nm-> batch_size, 1,
						&Dalpha,
						(double *)cur_params->b, cur_params->C_out,
						(double *)nm->one_vec, 1,
						&Dalpha,
						(double *)nm->layer_input[i + 1], cur_params->C_out));
		}
		if (cur_params->activation_mode != ACTIVATION_NONE) {
			//Replacing cuDNN call for Relu activation to custom Leaky Relu call
			//checkCUDNN(hipdnnActivationForward(nm->cudnn_handle, cur_params->actv_desc,&alpha,cur_params->output_tensor, 
nm->layer_input[i + 1],&beta,cur_params->output_tensor, nm->layer_input[i + 1]));
			activate_array_gpu((float *)nm->layer_input[i + 1], nm->layer_input_size[i+ 1], nm->stream_compute);
			
			
		}
		// std::cout << "FChere" << i << std::endl;
	}

	else if (nm->layer_type[i] == DROPOUT) {
		// std::cout << "Dropout\n";
		DropoutLayerParams *cur_params = (DropoutLayerParams *)nm->params[i];
		checkCUDNN(cudnnDropoutForward(nm->cudnn_handle, cur_params->dropout_desc,
					cur_params->input_tensor, nm->layer_input[i],
					cur_params->input_tensor, nm->layer_input[i + 1],
					cur_params->reserved_space,
					cur_params->reserved_space_size));
	}
	else if (nm->layer_type[i] == BATCHNORM) {
		// std::cout << "Batchnorm\n";
		BatchNormLayerParams *cur_params = (BatchNormLayerParams *)nm->params[i];

		checkCUDNN(hipdnnBatchNormalizationForwardInference(nm->cudnn_handle, cur_params->mode,
					&alpha, &beta,
					cur_params->input_tensor, nm->layer_input[i], cur_params->input_tensor, nm->layer_input[i+1], 
cur_params->sbmv_desc,
                                        cur_params->scale, cur_params->bias,
					cur_params->running_mean, cur_params->running_variance,
					cur_params->epsilon));

	}
	else if (nm->layer_type[i] == POOLING) {
		// std::cout << "Pooling\n";
		PoolingLayerParams *cur_params = (PoolingLayerParams *)nm->params[i];
		checkCUDNN(hipdnnPoolingForward(nm->cudnn_handle, cur_params->pool_desc,
					&alpha,
					cur_params->input_tensor, nm->layer_input[i],
					&beta,
					cur_params->output_tensor, nm->layer_input[i + 1]));
	}
	else if (nm->layer_type[i] == ACTV) {
		ActivationLayerParams *cur_params = (ActivationLayerParams *)nm->params[i];
		checkCUDNN(hipdnnActivationForward(nm->cudnn_handle, cur_params->actv_desc,
					&alpha,
					cur_params->input_tensor, nm->layer_input[i],
					&beta,
					cur_params->input_tensor, nm->layer_input[i + 1]));
	}
	else if (nm->layer_type[i] == SOFTMAX) {
		SoftmaxLayerParams *cur_params = (SoftmaxLayerParams *)nm->params[i];
		checkCUDNN(hipdnnSoftmaxForward(nm->cudnn_handle, cur_params->algo, cur_params->mode,
					&alpha,
					cur_params->input_tensor, nm->layer_input[i],
					&beta,
					cur_params->input_tensor, nm->layer_input[i + 1]));
		//-Copy the result produced by softmax layer from GPU to CPU

		checkCudaErrors(hipStreamSynchronize(nm->stream_compute)); /////-----check....
		float *result=(float *)malloc(nm->layer_input_size[i+1]*sizeof(float));
		checkCudaErrors(hipMemcpy(result, nm->layer_input[i+1], nm->layer_input_size[i+1]*sizeof(float), 
hipMemcpyDeviceToHost));

		//Infer the output class
	//	int *correct_count=0;
	//	nm->compareOutputCorrect(correct_count,nm->y);
	//	checkCNMEM(cnmemFree(nm->layer_input[nm->num_layers - 1], NULL));
	//	space_tracker.updateSpace(CnmemSpace::ADD, nm->layer_input_size[nm->num_layers - 1] * nm->data_type_size);
	//--
		int top=5;
		list *options=read_data_cfg("data/imagenet1k.data");//specify name of the file
		char *name_list = option_find_str(options, "names", 0);
    		if(!name_list) name_list = option_find_str(options, "labels", "data/labels.list");
    		if(top == 0) top = option_find_int(options, "top", 1);

	    int ii = 0;
	    char **names = get_labels(name_list);
	// clock_t time;
	    int *indexes = (int *)calloc(top, sizeof(int));
       // time=clock();
        top_k(result, nm->layer_input_size[i+1], top, indexes);//check parameters of this function
       // fprintf(stderr, "%s: Predicted in %f seconds.\n", input, sec(clock()-time));
        for(ii = 0; ii < top; ++ii){
            int index = indexes[ii];
            //if(net->hierarchy) printf("%d, %s: %f, parent: %s \n",index, names[index], predictions[index], 
(net->hierarchy->parent[index] >= 0) ? names[net->hierarchy->parent[index]] : "Root");
            //else printf("%s: %f\n",names[index], predictions[index]);
            //printf("index is %d: %5.2f%%: %s\n",index, result[index]*100, names[index]);
            printf("index is %d: %s\n",index, names[index]);
    		}
	}
	if (nm->layer_type[i] == CONV) {
		nm->lockedcnmemFree(cur_workspace, NULL);
		space_tracker.updateSpace(CnmemSpace::ADD, cur_workspace_size);
	}
	
	checkCudaErrors(hipStreamSynchronize(nm->stream_compute));
	//free the memory allocated to layer_input[i]
	nm->lockedcnmemFree(nm->layer_input[i], NULL);
	space_tracker.updateSpace(CnmemSpace::ADD, nm->layer_input_size[i] * nm->data_type_size);
			
}
int done=0;
pthread_cond_t Queue_Not_Empty=PTHREAD_COND_INITIALIZER; pthread_mutex_t lock =PTHREAD_MUTEX_INITIALIZER;
FILE *fpcp;

void ScheduleEngine::schedule(){
	//loops over all elements of prioriy Queue and dispatch all operations on GPU
	printf("Scheduling loop started\n");
	Operation tp;
	hipEvent_t global_start;
	hipEventCreate(&global_start);
	hipEventRecord(global_start);
	pthread_t tid;
        fpcp=fopen("newr.txt","w");

	pthread_create(tid,NULL, threadFunc,NULL);
	while(!Q.empty()){
		//pop element from queue
		tp=dequeue();
		//create events
		hipEventCreate(&tp.startop);
		hipEventCreate(&tp.endop);
		printf("Dispatching %d layer operation on GPU\n",tp.op_layer);
		hipEventRecord(tp.startop, (tp.model)->stream_compute);
		dispatch(&tp);
		hipEventRecord(tp.endop, (tp.model)->stream_compute);
		printf("Operation of %d layer completed on GPU\n",tp.op_layer);
		//add the operation object in timeQ queue;
		pthread_mutex_lock(&lock);
		timeQ.push(tp);
		pthread_mutex_unlock(&lock);
		pthread_cond_signal(&Queue_Not_Empty);
	}
	done=1;
	pthread_join(tid,NULL);
	fclose(fpcp);
}
	
void threadFunc(){
	Operation tp;

       for(;;){
		if(done==1) break;
	
		pthread_mutex_lock(&mVar);
		if(tempQ.empty())
			pthread_cond_wait(&Queue_Not_Empty,&lock);

		tp=timeQ.front();
		timeQ.pop();
		pthread_mutex_unlock(&lock);

		hipEventSynchronize(tp.startop);
		hipEventElapsedTime(&tp.time_to_start, global_start,tp.startop);
		hipEventSynchronize(tp.endop);
		hipEventElapsedTime(&tp.time_to_execute, tp.startop,tp.endop);
		fprintf(fpcf,"%d:%d:%f:%f \n", tp.pipeline,tp.op_layer,tp.time_to_start,tp.time_to_execute);
	}
}	

