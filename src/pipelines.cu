#include "hip/hip_runtime.h"
#include "solver.h"
#include <ScheduleEngine.h>
#include <cmath>
#include <cstdlib>
#include <dag.h>
#include <dirent.h>
#include <iostream>
#include <lsf_pipelines.h>
#include <lsf_scheduler.h>
#include <thread>
#include <utilities.h>

enum PROGRAM_TYPE {
    SEQUENTIAL = 1,
    COSCHEDULING = 2,
    LSF = 3,
    SMT = 4,
    LSF_PYTHON = 5
};

int str_ends_with(const char *s, const char *suffix) {
    size_t slen = strlen(s);
    size_t suffix_len = strlen(suffix);

    return suffix_len <= slen && !strcmp(s + slen - suffix_len, suffix);
}

// Task setup
int nos_of_tasks = 2;
int jobs_per_task[] = {2, 1};

typedef unsigned char uchar;

int pipelines = 0;
int numOfImages = 10;

int main(int argc, char *argv[]) {
    vector<NeuralNet> networks;
    if (argc != 6) {
        printf("Usage: build/pipeline.out data/weightfile1 data/weightfile2 "
               "images01/ images02 Program_type\n");
        printf("Program_Type 1:Sequential scheduling 2:Coscheduling lookup "
               "table 3:LSF scheduling 4:SMT schedule execution\n");
        exit(1);
    }
    vDNNConvAlgo vdnn_conv_algo = vDNN_MEMORY_OPTIMAL;
    vDNNType vdnn_type = vDNN_ALL;
    int batch_size = 1;
    long long dropout_seed = 1;
    float softmax_eps = 1e-8;
    float init_std_dev = 0.1;
    /*	NeuralNet vggnet(layer_specifier, DATA_FLOAT, batch_size, TENSOR_NCHW,
       dropout_seed, softmax_eps, init_std_dev, vdnn_type, vdnn_conv_algo, SGD);


            NeuralNet alexnet(layer_specifier1, DATA_FLOAT, batch_size,
       TENSOR_NCHW, dropout_seed, softmax_eps, init_std_dev, vdnn_type,
       vdnn_conv_algo, SGD);

     */

    pipelines++;
    vector<LayerSpecifier> layer_specifier3;
    // conv1
    {
        ConvDescriptor conv1;
        conv1.initializeValues(3, 16, 3, 3, 16, 16, 1, 1, 1, 1, 1, RELU);
        // conv1.initializeValues(3, 16, 3, 3, 224,224, 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv1;
        layer_specifier3.push_back(temp);
    }
    // max pool 1
    {
        PoolingDescriptor poo01;
        poo01.initializeValues(16, 2, 2, 16, 16, 0, 0, 2, 2, POOLING_MAX);
        // poo01.initializeValues(16, 2, 2, 224,224 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo01;
        layer_specifier3.push_back(temp);
    }
    // conv2
    {
        ConvDescriptor conv2;
        conv2.initializeValues(16, 32, 3, 3, 8, 8, 1, 1, 1, 1, 1, RELU);
        // conv2.initializeValues(16, 32, 3, 3, 112, 112, 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv2;
        layer_specifier3.push_back(temp);
    }
    // max pool 2
    {
        PoolingDescriptor poo02;
        poo02.initializeValues(32, 2, 2, 8, 8, 0, 0, 2, 2, POOLING_MAX);
        // poo02.initializeValues(32, 2, 2, 112,112 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo02;
        layer_specifier3.push_back(temp);
    }
    // conv3
    {
        ConvDescriptor conv3;
        conv3.initializeValues(32, 64, 3, 3, 4, 4, 1, 1, 1, 1, 1, RELU);
        // conv3.initializeValues(32, 64 , 3, 3, 56,56, 1, 1 , 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv3;
        layer_specifier3.push_back(temp);
    }
    // max pool 3
    {
        PoolingDescriptor poo03;
        poo03.initializeValues(64, 2, 2, 4, 4, 0, 0, 2, 2, POOLING_MAX);
        // poo03.initializeValues(64, 2, 2, 56,56 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo03;
        layer_specifier3.push_back(temp);
    }
    // conv4
    {
        ConvDescriptor conv4;
        conv4.initializeValues(16, 128, 3, 3, 2, 2, 1, 1, 1, 1, 1, RELU);
        // conv4.initializeValues(16,128, 3, 3, 28,28, 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv4;
        layer_specifier3.push_back(temp);
    }
    // max pool 4
    {
        PoolingDescriptor poo04;
        poo04.initializeValues(128, 2, 2, 2, 2, 0, 0, 1, 1, POOLING_MAX);
        // poo04.initializeValues(128 , 2, 2, 28, 28 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo04;
        layer_specifier3.push_back(temp);
    }
    // conv5
    {
        ConvDescriptor conv5;
        conv5.initializeValues(128, 128, 3, 3, 2, 2, 1, 1, 1, 1, 1, RELU);
        // conv5.initializeValues(128,128 ,3,3, 14, 14 , 1,1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv5;
        layer_specifier3.push_back(temp);
    }
    // max pool 5
    {
        PoolingDescriptor poo05;
        poo05.initializeValues(128, 2, 2, 2, 2, 0, 0, 1, 1, POOLING_MAX);
        // poo05.initializeValues(128 , 2, 2, 14,14 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo05;
        layer_specifier3.push_back(temp);
    }
    // conv6
    {
        ConvDescriptor conv6;
        conv6.initializeValues(128, 256, 3, 3, 2, 2, 1, 1, 1, 1, 1, RELU);
        // conv6.initializeValues( 128,256,3,3, 7,7 , 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv6;
        layer_specifier3.push_back(temp);
    }
    // conv7
    {
        ConvDescriptor conv7;
        conv7.initializeValues(256, 125, 1, 1, 2, 2, 1, 1, 1, 1, 1, SIGMOID);
        // conv7.initializeValues( 256 , 125, 1, 1, 7,7 , 0, 0, 1, 1, 1,RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv7;
        layer_specifier3.push_back(temp);
    }
    // Region layer
    {
        RegionDescriptor region1;
        // region1.initializeValues(channels,w,h,nuw,classes,coords);
        region1.initializeValues(125, 2, 2, 5, 20, 4);
        // region1.initializeValues(125,7,7,5,20,4);
        LayerSpecifier temp;
        temp.initPointer(REGION);
        *((RegionDescriptor *)temp.params) = region1;
        layer_specifier3.push_back(temp);
    }

    networks.push_back(NeuralNet(layer_specifier3, DATA_FLOAT, batch_size,
                                 TENSOR_NCHW, dropout_seed, softmax_eps,
                                 init_std_dev, vdnn_type, vdnn_conv_algo, SGD,
                                 argv[1], argv[3]));

    pipelines++;
    vector<LayerSpecifier> layer_specifier2;
    // conv1
    {
        ConvDescriptor conv1;
        conv1.initializeValues(3, 16, 3, 3, 16, 16, 1, 1, 1, 1, 1, RELU);
        // conv1.initializeValues(3, 16, 3, 3, 416, 416, 1, 1, 1, 1, 1, RELU);
        // conv1.initializeValues(3, 16, 3, 3, 224, 224, 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv1;
        layer_specifier2.push_back(temp);
    }
    // max pool 1
    {
        PoolingDescriptor poo01;
        poo01.initializeValues(16, 2, 2, 16, 16, 0, 0, 2, 2, POOLING_MAX);
        // poo01.initializeValues(16, 2, 2, 416, 416 , 0, 0, 2, 2, POOLING_MAX);
        // poo01.initializeValues(16, 2, 2, 224,224 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo01;
        layer_specifier2.push_back(temp);
    }
    // conv2
    {
        ConvDescriptor conv2;
        conv2.initializeValues(16, 32, 3, 3, 8, 8, 1, 1, 1, 1, 1, RELU);
        // conv2.initializeValues(16, 32, 3, 3, 208, 208, 1, 1, 1, 1, 1, RELU);
        // conv2.initializeValues(16, 32, 3, 3, 112, 112, 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv2;
        layer_specifier2.push_back(temp);
    }
    // max pool 2
    {
        PoolingDescriptor poo02;
        poo02.initializeValues(32, 2, 2, 8, 8, 0, 0, 2, 2, POOLING_MAX);
        // poo02.initializeValues(32, 2, 2, 208, 208 , 0, 0, 2, 2, POOLING_MAX);
        // poo02.initializeValues(32, 2, 2, 112, 112, 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo02;
        layer_specifier2.push_back(temp);
    }
    // conv3
    {
        ConvDescriptor conv3;
        conv3.initializeValues(32, 64, 3, 3, 4, 4, 1, 1, 1, 1, 1, RELU);
        // conv3.initializeValues(32, 64 , 3, 3, 104, 104, 1, 1 , 1, 1, 1,
        // RELU); conv3.initializeValues(32, 64 , 3, 3, 56, 56, 1, 1 , 1, 1, 1,
        // RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv3;
        layer_specifier2.push_back(temp);
    }
    // max pool 3
    {
        PoolingDescriptor poo03;
        poo03.initializeValues(64, 2, 2, 4, 4, 0, 0, 2, 2, POOLING_MAX);
        // poo03.initializeValues(64, 2, 2, 104, 104 , 0, 0, 2, 2, POOLING_MAX);
        // poo03.initializeValues(64, 2, 2, 56, 56 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo03;
        layer_specifier2.push_back(temp);
    }

    // conv4
    {
        ConvDescriptor conv4;
        // conv4.initializeValues(64,128, 3, 3, 2,2, 1, 1, 1, 1, 1, RELU);
        conv4.initializeValues(64, 64, 3, 3, 2, 2, 1, 1, 1, 1, 1, RELU);
        // conv4.initializeValues(16,128, 3, 3, 52, 52, 1, 1, 1, 1, 1, RELU);
        // conv4.initializeValues(16,128, 3, 3, 28,28, 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv4;
        layer_specifier2.push_back(temp);
    }
    // max pool 4
    {
        PoolingDescriptor poo04;
        // poo04.initializeValues(128 , 2, 2, 2, 2 , 0, 0, 1, 1, POOLING_MAX);
        poo04.initializeValues(64, 2, 2, 2, 2, 0, 0, 1, 1, POOLING_MAX);
        // poo04.initializeValues(128 , 2, 2, 52, 52 , 0, 0, 2, 2, POOLING_MAX);
        // poo04.initializeValues(128 , 2, 2, 28, 28, 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo04;
        layer_specifier2.push_back(temp);
    }
    // conv5
    {
        ConvDescriptor conv5;
        //--conv5.initializeValues(128,128 , 3, 3, 4,4, 1,1,1, 1, 1, RELU);
        // conv5.initializeValues(128,256 , 3, 3, 2,2, 1,1,1, 1, 1, RELU);
        conv5.initializeValues(64, 128, 3, 3, 2, 2, 1, 1, 1, 1, 1, RELU);
        // conv5.initializeValues(128,128 , 3, 3, 26, 26, 1,1,1, 1, 1, RELU);
        // conv5.initializeValues(128,128 , 3, 3, 14, 14, 1,1,1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv5;
        layer_specifier2.push_back(temp);
    }
    // max pool 5
    {
        PoolingDescriptor poo05;
        // poo05.initializeValues(256 , 2, 2, 2,2 , 0, 0, 1, 1, POOLING_MAX);
        poo05.initializeValues(128, 2, 2, 2, 2, 0, 0, 1, 1, POOLING_MAX);
        // poo05.initializeValues(128 , 2, 2, 26, 26 , 0, 0, 2, 2, POOLING_MAX);
        // poo05.initializeValues(128 , 2, 2, 14,14 , 0, 0, 2, 2, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo05;
        layer_specifier2.push_back(temp);
    }
    // conv6
    {
        ConvDescriptor conv6;
        //--conv6.initializeValues( 128,256,3,3, 2,2, 1, 1, 1, 1, 1, RELU);
        // conv6.initializeValues( 256,512,3,3, 2,2, 1, 1, 1, 1, 1, RELU);
        conv6.initializeValues(128, 256, 3, 3, 2, 2, 1, 1, 1, 1, 1, RELU);
        // conv6.initializeValues( 128,256,3,3, 13,13, 1, 1, 1, 1, 1, RELU);
        // conv6.initializeValues( 128,256,3,3, 7,7, 1, 1, 1, 1, 1, RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv6;
        layer_specifier2.push_back(temp);
    }
    // max pool 6
    {
        PoolingDescriptor poo06;
        //--poo06.initializeValues(256 , 2, 2, 2,2 ,0, 0, 2, 2, POOLING_MAX);
        // poo06.initializeValues(512 , 2, 2, 2,2 ,0, 0, 2, 2, POOLING_MAX);
        poo06.initializeValues(256, 2, 2, 2, 2, 0, 0, 2, 2, POOLING_MAX);
        // poo06.initializeValues(256 , 2, 2, 13, 13 ,0, 0, 1, 1, POOLING_MAX);
        // poo06.initializeValues(256 , 2, 2, 7, 7 ,0, 0, 1, 1, POOLING_MAX);
        LayerSpecifier temp;
        temp.initPointer(POOLING);
        *((PoolingDescriptor *)temp.params) = poo06;
        layer_specifier2.push_back(temp);
    }

    // conv7
    {
        ConvDescriptor conv7;
        // conv7.initializeValues( 512 , 1024, 3, 3, 1,1, 1, 1, 1, 1, 1,RELU);
        conv7.initializeValues(256, 512, 3, 3, 1, 1, 1, 1, 1, 1, 1, RELU);
        // conv7.initializeValues( 256 , 1024, 3, 3, 12, 12, 1, 1, 1, 1,
        // 1,RELU); conv7.initializeValues( 256 , 1024, 3, 3, 6, 6, 1, 1, 1, 1,
        // 1,RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv7;
        layer_specifier2.push_back(temp);
    }
    // conv8
    {
        ConvDescriptor conv8;
        //--conv8.initializeValues( 256 , 1024, 3, 3, 1,1, 1, 1, 1, 1, 1,RELU);
        // conv8.initializeValues( 1024 , 1024, 3, 3, 1,1, 1, 1, 1, 1, 1,RELU);
        conv8.initializeValues(512, 512, 3, 3, 1, 1, 1, 1, 1, 1, 1, RELU);
        // conv8.initializeValues( 1024 , 1024, 3, 3, 12, 12, 1, 1, 1, 1,
        // 1,RELU); conv8.initializeValues( 1024 , 1024, 3, 3, 6,6, 1, 1, 1, 1,
        // 1,RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv8;
        layer_specifier2.push_back(temp);
    }
    // conv9
    {
        ConvDescriptor conv9;
        //-conv9.initializeValues( 256 , 1024, 3, 3, 1,1, 1, 1, 1, 1, 1,RELU);
        // conv9.initializeValues( 1024 , 125, 3, 3, 1,1, 1, 1, 1, 1, 1,RELU);
        conv9.initializeValues(512, 125, 3, 3, 1, 1, 1, 1, 1, 1, 1, RELU);
        // conv9.initializeValues( 1024 , 125, 1, 1, 12, 12, 1, 1, 1, 1,
        // 1,RELU); conv9.initializeValues( 1024 , 125, 1, 1, 6, 6, 1, 1, 1, 1,
        // 1,RELU);
        LayerSpecifier temp;
        temp.initPointer(CONV);
        *((ConvDescriptor *)temp.params) = conv9;
        layer_specifier2.push_back(temp);
    }
    // Region layer
    {
        RegionDescriptor region1;
        // region1.initializeValues(channels,w,h,nuw,classes,coords);
        region1.initializeValues(125, 1, 1, 5, 20, 4);
        // region1.initializeValues(125,6,6,5,20,4);
        LayerSpecifier temp;
        temp.initPointer(REGION);
        *((RegionDescriptor *)temp.params) = region1;
        layer_specifier2.push_back(temp);
    }

    // cost layer as per specification

    networks.push_back(NeuralNet(layer_specifier2, DATA_FLOAT, batch_size,
                                 TENSOR_NCHW, dropout_seed, softmax_eps,
                                 init_std_dev, vdnn_type, vdnn_conv_algo, SGD,
                                 argv[2], argv[4]));

    // Now inference will start: Operations of each layer will be scheduled by
    // ScheduleEngine

    // Create an object of ScheduleEngine
    ScheduleEngine se;
    /* se.model1=&networks[0];
    se.model2=&networks[1]; */

    // warmup code starts here
    //--------------------
    for (auto &network : networks) {
        auto zerothLayer = new InputOperation("kite.jpg", &network, 0, 'm', 0);
        createLinearDAG(zerothLayer);
        printf("Starting Warm up code\n");
        Operation *currentOperation = zerothLayer;
        while (currentOperation != nullptr) {
            se.enqueue(currentOperation);
            currentOperation = currentOperation->children.back();
        }
        se.warmup_schedule(zerothLayer);
        printf("Warming up code execution completed\n");
        network.cur_prefetch_layer = 0;
        fseek(network.wfp, 0, SEEK_SET);
        destroyLinearDAG(&zerothLayer);
    }
    // warmup code ends here

    DIR *d1, *d2;
    struct dirent *dir;
    char **list1, **list2;
    int i = 0;
    float ms = 0;
    float total_time = 0;
    list1 = (char **)malloc(numOfImages * sizeof(char *));
    list2 = (char **)malloc(numOfImages * sizeof(char *));
    printf("P1: %s", networks[0].imgpath);
    printf("P2: %s", networks[1].imgpath);
    d1 = opendir(networks[0].imgpath); // arg[3] for pipeline01  image path
    d2 = opendir(networks[1].imgpath); // arg[4] for pipeline01  image path
    if (d1) {
        while ((dir = readdir(d1)) != NULL && i < numOfImages) {

            if (!strcmp(dir->d_name, "."))
                continue;
            if (!strcmp(dir->d_name, ".."))
                continue;
            if (str_ends_with(dir->d_name, ".JPEG")) {
                list1[i] =
                    (char *)malloc((strlen(dir->d_name) + 1) * sizeof(char));
                strcpy(list1[i], dir->d_name);
                printf("%s\n", list1[i]);
                i++;
            }
        }
        closedir(d1);
    } else {
        printf("Unable to open directory %s\n", networks[0].imgpath);
        exit(1);
    }

    i = 0;

    if (d2) {
        while ((dir = readdir(d2)) != NULL && i < numOfImages) {

            if (!strcmp(dir->d_name, "."))
                continue;
            if (!strcmp(dir->d_name, ".."))
                continue;
            if (str_ends_with(dir->d_name, ".JPEG")) {
                list2[i] =
                    (char *)malloc((strlen(dir->d_name) + 1) * sizeof(char));
                strcpy(list2[i], dir->d_name);
                printf("%s\n", list2[i]);
                i++;
            }
        }

        closedir(d2);
    } else {
        printf("Unable to open directory %s\n", networks[1].imgpath);
        exit(1);
    }
    assert(strlen(argv[argc - 1]) == 1);

    switch (atoi(argv[argc - 1])) {
    case PROGRAM_TYPE::SEQUENTIAL: {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        //--------------
        // Create global event
        se.createGlobalEvent();
        FILE *fpcf = fopen("output/stats_mem_seq.txt", "w");
        char filename[100];
        for (int no = 0; no < numOfImages; no++) {
            strcpy(filename, networks[0].imgpath);
            strcat(filename, list1[no]);
            auto zerothLayer1 =
                new InputOperation(filename, &networks[0], 0, 'm', 1);
            createLinearDAG(zerothLayer1);
            hipEventRecord(start);
            se.schedule_sequential(zerothLayer1, fpcf);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&ms, start, stop);
            total_time += ms;

            // execution of pipeline 01 is over, so deallocate its space
            // networks[0].deallocateSpace();
            strcpy(filename, networks[1].imgpath);
            strcat(filename, list2[no]);
            auto zerothLayer2 =
                new InputOperation(filename, &networks[1], 0, 'm', 2);
            createLinearDAG(zerothLayer2);
            hipEventRecord(start);
            se.schedule_sequential(zerothLayer2, fpcf);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&ms, start, stop);
            total_time += ms;
        }
        fclose(fpcf);
        printf("Total time to process %d images is %f\n", numOfImages,
               total_time);
        // networks[1].deallocateSpace();
        break;
    }
    case PROGRAM_TYPE::COSCHEDULING: {
        vector<Operation *> pipe1, pipe2;
        // Parallel execution of two pipelines
        auto zerothLayer1 =
            new InputOperation("data/dog.jpg", &networks[0], 0, 'm', 1);
        createLinearDAG(zerothLayer1);
        Operation *currentOperation = zerothLayer1;
        while (currentOperation != nullptr) {
            printf("%c  ", currentOperation->op_type);
            if (currentOperation->op_type == 'c') {
                pipe1.push_back(currentOperation);
            }
            currentOperation = currentOperation->children.back();
        }
        printf("\n");
        auto zerothLayer2 =
            new InputOperation("data/eagle.jpg", &networks[1], 0, 'm', 2);
        createLinearDAG(zerothLayer2);
        currentOperation = zerothLayer2;
        while (currentOperation != nullptr) {
            printf("%c  ", currentOperation->op_type);
            if (currentOperation->op_type == 'c') {
                pipe2.push_back(currentOperation);
            }
            currentOperation = currentOperation->children.back();
        }
        printf("\nNumber of elements in pipe1 is %ld\n", pipe1.size());
        printf("Number of elements in pipe1 is %ld\n", pipe2.size());
        // call schedule profile function
        printf("starting execution of schedule_profile for generating "
               "co-scheduling table\n");
        se.schedule_profile(zerothLayer1, zerothLayer2, pipe1, pipe2);
        printf("Finished preparing co-scheduling table");
        break;
    }
    case PROGRAM_TYPE::LSF: {
        ifstream timingFile;
        timingFile.open("output/arrival-execution.txt");
        string filename = networks[0].imgpath;
        filename += list1[0];
        auto zerothLayer1 =
            new InputOperation(filename, &networks[0], 0, 'm', 0);
        createLinearDAG(zerothLayer1);
        filename = networks[1].imgpath;
        filename += list2[0];
        auto zerothLayer2 =
            new InputOperation(filename, &networks[1], 0, 'm', 1);
        createLinearDAG(zerothLayer2);
        // loadTimings(timingFile1, zerothLayer1);
        fillExecutionTime(timingFile, {zerothLayer1, zerothLayer2});
        // Start the execution of LSF
        vector<InputOperation *> v;
        v.push_back(zerothLayer1);
        v.push_back(zerothLayer2);
        start(v);
        break;
    }
    case PROGRAM_TYPE::SMT: {
        ScheduleEngine se;
        vector<InputOperation *> zerothLayer;
        ifstream timingFile;
        timingFile.open("output/smt-arrival-stream.txt");
        for (int i = 0; i < jobs_per_task[0]; i++) {
            string filename = networks[0].imgpath;
            filename += list1[i];
            zerothLayer.push_back(
                new InputOperation(filename, &networks[0], 0, 'm', 0));
            createLinearDAG(zerothLayer.back());
        }
        for (int i = 0; i < jobs_per_task[1]; i++) {
            string filename = networks[1].imgpath;
            filename += list2[i];
            zerothLayer.push_back(
                new InputOperation(filename, &networks[1], 0, 'm', 1));
            createLinearDAG(zerothLayer.back());
        }
        fillSMTDetails(timingFile, zerothLayer);
        std::priority_queue<Operation *, std::vector<Operation *>,
                            compareStartTimings>
            operationQueue;
        for (auto &zeroLayer : zerothLayer) {
            dagToPriorityQueue(operationQueue, (Operation *)zeroLayer);
        }
        chrono::time_point<chrono::steady_clock> timeGlobalStart =
            chrono::steady_clock::now(); // globalStart
        while (!operationQueue.empty()) {
            auto currentOperation = operationQueue.top();
            operationQueue.pop();
            {
                if (currentOperation->op_type == 'c') {
                    hipEventSynchronize(
                        currentOperation->parents.back()->endop);
                    auto currentStream =
                        (currentOperation->chosenStream == 'H')
                            ? ScheduleEngine::HIGH_COMPUTE_STREAM
                            : ScheduleEngine::LOW_COMPUTE_STREAM;
                    checkCudaErrors(
                        hipEventRecord(currentOperation->startop,
                                        se.compute_streams[currentStream]));
                    assert(currentOperation->parents.back()->op_type == 'm');
                    se.dispatch(currentOperation, currentStream);
                    checkCudaErrors(
                        hipEventRecord(currentOperation->endop,
                                        se.compute_streams[currentStream]));
                } else if (currentOperation->op_type == 'm') {
                    checkCudaErrors(hipEventRecord(currentOperation->endop,
                                                    se.memoryStream));
                    if (currentOperation->op_layer == 0) {
                        InputOperation *zerothLayer =
                            static_cast<InputOperation *>(currentOperation);
                        zerothLayer->model->loadFile(
                            const_cast<char *>((zerothLayer->filename).c_str()),
                            se.memoryStream);
                    } else {
                        currentOperation->model->prefetchWeights(
                            currentOperation->op_layer - 1,
                            se.memoryStream); //-1 missing here
                    }
                    checkCudaErrors(hipEventRecord(currentOperation->endop,
                                                    se.memoryStream));
                }
            }
            // sleep for (duration = start time of next op - start time of
            // current op)
            std::this_thread::sleep_for(
                std::chrono::duration<double, std::milli>(
                    operationQueue.top()->time_to_start -
                    currentOperation->time_to_start));
        }
        cout << "Total time for processing = "
             << (chrono::duration_cast<chrono::duration<double, std::milli>>(
                     chrono::steady_clock::now() - timeGlobalStart))
                    .count()
             << endl;
        break;
    }

    case PROGRAM_TYPE::LSF_PYTHON: {
        lsf_initialize();
        vector<InputOperation *> zerothLayer;
        ifstream timingFile;
        timingFile.open("output/lsf-arrival-stream.txt");
        for (int i = 0; i < jobs_per_task[0]; i++) {
            string filename = networks[0].imgpath;
            filename += list1[i];
            zerothLayer.push_back(
                new InputOperation(filename, &networks[0], 0, 'm', 0));
            createLinearDAG(zerothLayer.back());
        }
        for (int i = 0; i < jobs_per_task[1]; i++) {
            string filename = networks[1].imgpath;
            filename += list2[i];
            zerothLayer.push_back(
                new InputOperation(filename, &networks[1], 0, 'm', 1));
            createLinearDAG(zerothLayer.back());
        }
        fillSMTDetails(timingFile, zerothLayer);
        std::priority_queue<Operation *, std::vector<Operation *>,
                            compareStartTimings>
            operationQueue;
        for (auto &zeroLayer : zerothLayer) {
            dagToPriorityQueue(operationQueue, (Operation *)zeroLayer);
        }
        chrono::time_point<chrono::steady_clock> timeGlobalStart =
            chrono::steady_clock::now(); // globalStart
        while (!operationQueue.empty()) {
            auto currentOperation = operationQueue.top();
            operationQueue.pop();
            {
                auto currentStream = atoi(string(1, currentOperation->chosenStream).c_str());
                if (currentOperation->op_type == 'c') {
                    hipEventSynchronize(
                        currentOperation->parents.back()->endop);
                    checkCudaErrors(hipEventRecord(currentOperation->startop,
                                                    lsf_stream[currentStream]));
                    assert(currentOperation->parents.back()->op_type == 'm');
                    lsf_dispatch(currentOperation, currentStream);
                    checkCudaErrors(hipEventRecord(currentOperation->endop,
                                                    lsf_stream[currentStream]));
                } else if (currentOperation->op_type == 'm') {
                    checkCudaErrors(hipEventRecord(currentOperation->endop,
                                                    lsf_stream[currentStream]));
                    if (currentOperation->op_layer == 0) {
                        InputOperation *zerothLayer =
                            static_cast<InputOperation *>(currentOperation);
                        zerothLayer->model->loadFile(
                            const_cast<char *>((zerothLayer->filename).c_str()),
                            lsf_stream[currentStream]);
                    } else {
                        currentOperation->model->prefetchWeights(
                            currentOperation->op_layer - 1,
                            lsf_stream[currentStream]);
                    }
                    checkCudaErrors(hipEventRecord(currentOperation->endop,
                                                    lsf_stream[currentStream]));
                }
            }
            // sleep for (duration = start time of next op - start time of
            // current op)
            std::this_thread::sleep_for(
                std::chrono::duration<double, std::milli>(
                    operationQueue.top()->time_to_start -
                    currentOperation->time_to_start));
        }
        cout << "Total time for processing = "
             << (chrono::duration_cast<chrono::duration<double, std::milli>>(
                     chrono::steady_clock::now() - timeGlobalStart))
                    .count()
             << endl;
        break;
    }
    }
}
/*

//--------------
char filename[100];
for (int no=0;no<numOfImages;no++)
{
        strcpy(filename,networks[0].imgpath);
        strcat(filename, list1[no]);
        printf("Image No: %d Processing  %s file by pipeline01\n",no,filename);
        //Load the image from the directory into pipeline01
        networks[0].loadFile(filename);

        //Parallel execution of two pipelines
        //printf("Adding operations of tiny Yolov1 in Queue\n");
        for(int i=0;i < networks[0].num_layers;i++){
                op=Operation();
                op.model=&networks[0]; op.op_layer=i;
                op.priority= i;
                op.pipeline=1;
                //se.enqueue(op);
                pipe1.push_back(op);
        }

//Uncomment this block for sequential executin
/*
        hipEventRecord(start);
        se.schedule_sequential(&networks[0]);
        //se.schedule();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start,stop);
        total_time+=ms;
*/
/*
        strcpy(filename,networks[1].imgpath);
        strcat(filename, list2[no]);
        printf("Processing  %s file by pipeline02\n",filename);
        //Load the image from the directory into pipeline02
        networks[1].loadFile(filename);
        for(int i=0;i < networks[1].num_layers;i++){
                op=Operation();
                op.model=&networks[1]; op.op_layer=i;//+networks[0].num_layers;
                op.priority= i;
                op.pipeline=2;
                //se.enqueue(op);
                pipe2.push_back(op);
        }


        //Call schedule routine

        hipEventRecord(start);
        se.schedule_profile(pipe1, pipe2); //comment for seqential execution
        //se.schedule_sequential(&networks[1]);//comment for parallel execution
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start,stop);
        total_time+=ms;

}
printf("Total time to process %d images is %f\n", numOfImages, total_time);
        networks[0].deallocateSpace();
        networks[1].deallocateSpace();
*/